#include "hip/hip_runtime.h"
#ifndef __HIPCC__
#define __HIPCC__
#endif
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include "hip/hip_runtime.h"
#include ""

#define N 1024
#define A_WIDTH N
#define A_HEIGHT N
#define B_WIDTH N
#define B_HEIGHT N
#define C_WIDTH B_WIDTH
#define C_HEIGHT A_HEIGHT

#define BLOCK_SIZE 8
#define NUM_SUBS (A_WIDTH / BLOCK_SIZE)

__device__ float d_A[A_HEIGHT][A_WIDTH];
__device__ float d_B[B_HEIGHT][B_WIDTH];
__device__ float d_C[C_HEIGHT][C_WIDTH];

float h_A[A_HEIGHT][A_WIDTH];
float h_B[B_HEIGHT][B_WIDTH];
float h_C[C_HEIGHT][C_WIDTH];
float h_C_ref[C_HEIGHT][C_WIDTH];

void checkCUDAError(const char *msg);
void matrixMulCPU(float A[A_HEIGHT][A_WIDTH], float B[B_HEIGHT][B_WIDTH], float C[C_HEIGHT][C_WIDTH]);
int matrixMulTest(float C[C_HEIGHT][C_WIDTH], float Cref[C_HEIGHT][C_WIDTH]);

printf("Using thread block size = %d \n", BLOCK_SIZE);

__global__ void matrixMulCUDA()
{
    // Block index
	int blockCol = blockIdx.x;
	int blockRow = blockIdx.y;
	int threadCol = threadIdx.x;
	int threadRow = threadIdx.y;
	int col = blockCol*BLOCK_SIZE + threadCol;
	int row = blockRow*BLOCK_SIZE + threadRow;
    
	float Csub = 0;
	//iterate A_WIDTH (same as B_HEIGHT) to calculate the product
	for (int k = 0; k < A_WIDTH; k++){
		Csub += d_A[row][k] * d_B[k][col]; 
	}

	// Store the product value of C matrix
	d_C[row][col] = Csub;
}


__global__ void matrixMulCUDASharedMemory()
{
    //Define some shared memory for a sub block of matrices A an B
    __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
	__shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];
    
	// Block index
	int blockCol = blockIdx.x;
	int blockRow = blockIdx.y;
	int threadCol = threadIdx.x;
	int threadRow = threadIdx.y;

    //Running sum of product of A and B matrices
    float Csub = 0;
 
	//iterate through the number of sub matrices of A and B
	for (int i = 0; i < NUM_SUBS; i++){
		//Indices of A and B matrix required to load the shared block of memory
		int a_col = threadCol + i*BLOCK_SIZE; 
		int a_row = threadRow + blockRow*BLOCK_SIZE;
		int b_col = threadCol + blockCol*BLOCK_SIZE;
		int b_row = threadRow + i*BLOCK_SIZE; 
		        
        //Each thread should load a single element of sub block of matrices A an B into shared memory
		//global indices along blocks 
		int col =  threadCol;
		int row =  threadRow; 
		As[row][col] = d_A[a_row][a_col];
		Bs[row][col] = d_B[b_row][b_col];

        // Sync to ensure sub matrix is fully loaded
		__syncthreads();

		        
        // Sum products of A and B sub matrices
		for (int k = 0; k < BLOCK_SIZE; ++k)
		{
			Csub += As[row][k] * Bs[k][col];
		}
        
        // Sync to prevent run ahead (blocks loading new SM values before others have completed)
		__syncthreads();    
	}

    //Calculate the indices of sub matrix C
	int c_col = threadCol + blockCol*BLOCK_SIZE; //==b_col;
	int c_row = threadRow + blockRow*BLOCK_SIZE; //==a_row;
    
	// Store the product value of C matrix
	d_C[c_row][c_col] = Csub;
}


int main(int argc, char **argv)

{
	unsigned int mem_size_A, mem_size_B, mem_size_C;
	unsigned int col, row, errors;
	int maxActiveBlocks;
	float msec, occupancy;
	int nDevice;
	int count;
	hipDeviceProp_t prop;


	hipGetDeviceCount(&nDevice);
	for (count = 0; count < nDevice; count++){
		hipGetDeviceProperties (&prop, count);
		if (count == 0){
			printf("Properties of Device %d \n", count);
			printf("maxBlocksPerMultiProcessor = %d \n", prop.maxBlocksPerMultiProcessor);
			printf("maxThreadsPerMultiProcessor = %d \n", prop.maxThreadsPerMultiProcessor);
			printf("multiProcessorCount = %d \n", prop.multiProcessorCount);
		}
	}

	hipEvent_t start, stop; 

	if (A_WIDTH != B_HEIGHT){
		printf("Error: A_HEIGHT and B_WIDTH do not match\n");
	}

	mem_size_A = sizeof(float)* A_WIDTH* A_HEIGHT;
	mem_size_B = sizeof(float)* B_WIDTH* B_HEIGHT;
	mem_size_C = sizeof(float)* C_WIDTH* C_HEIGHT;

	// Initialise A
	for (row = 0; row < A_HEIGHT; row++)
	for (col = 0; col <A_WIDTH; col++)
		h_A[row][col] = (float)rand() / RAND_MAX;
	// Initialise B
	for (row = 0; row < B_HEIGHT; row++)
	for (col = 0; col <B_WIDTH; col++)
		h_B[row][col] = (float)rand() / RAND_MAX;


	hipMemcpyToSymbol(HIP_SYMBOL(d_A), h_A, mem_size_A); // accessing global variables via the runtime API
	hipMemcpyToSymbol(HIP_SYMBOL(d_B), h_B, mem_size_B); //because variables d_A, d_B are statically defined (and allocated) in GPU global memory (with __device__ qualifier)
	checkCUDAError("CUDA memcpy");

	// Allocate CUDA events that we'll use for timing
	hipEventCreate(&start);
	hipEventCreate(&stop);
	checkCUDAError("CUDA event creation");

	// Setup execution parameters
	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid(C_WIDTH / BLOCK_SIZE, C_HEIGHT / BLOCK_SIZE);
	hipEventRecord(start);
	
    //Kernel
    //matrixMulCUDA << < grid, threads >> >();
    //Shared memory version of the kernel
    matrixMulCUDASharedMemory << < grid, threads >> >();
    
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	checkCUDAError("CUDA kernel execution and timing");

	hipEventElapsedTime(&msec, start, stop);
	hipDeviceSynchronize();
	checkCUDAError("CUDA timing");

	// Compute the ocupancy
	int ThreadsPerBlock = BLOCK_SIZE * BLOCK_SIZE;
	printf("test %d = %d \n", threads.x * threads.y * threads.z, ThreadsPerBlock );
	occupancy = prop.maxBlocksPerMultiProcessor * ThreadsPerBlock / (float)(prop.maxThreadsPerMultiProcessor * prop.multiProcessorCount);

	// Copy result from device to host
	hipMemcpyFromSymbol(h_C, HIP_SYMBOL(d_C), mem_size_C);
	checkCUDAError("CUDA memcpy results");

	// Compute reference CPU version
	matrixMulCPU(h_A, h_B, h_C_ref);

	// Check for errors
	errors = matrixMulTest(h_C, h_C_ref);
	if (errors)
		printf("%d total errors\n", errors);
	else
		printf("Test passed successfully\n");

	printf("Kernel time was %f with theoretical occupancy of %f \n", msec, occupancy);

}

void matrixMulCPU(float A[A_HEIGHT][A_WIDTH], float B[C_HEIGHT][C_WIDTH], float C[C_HEIGHT][C_WIDTH])
{
	int col, row, k;
	for (row = 0; row < C_HEIGHT; row++){
		for (col = 0; col < C_WIDTH; col++){
			C[row][col] = 0;
			for (k = 0; k < A_WIDTH; k++){
				C[row][col] += A[row][k] * B[k][col];
			}
		}
	}

}

int matrixMulTest(float C[C_HEIGHT][C_WIDTH], float Cref[C_HEIGHT][C_WIDTH])
{
	int errors = 0;
	int row, col;
	float epsilon = 1e+3; //precision difference between compilers. 
	//epsilon = 1e+3 for -fmad=false, see https://developer.download.nvidia.com/assets/cuda/files/NVIDIA-CUDA-Floating-Point.pdf

	for (row = 0; row < C_HEIGHT; row++){
		for (col = 0; col < C_WIDTH; col++){
			if (round(C[row][col]*epsilon) != round(Cref[row][col]*epsilon)){
				errors++;
				printf("Device item c[%d][%d] = %f does not match host result %f\n", row, col, C[row][col], Cref[row][col]);
			}
		}
	}
	return errors;
}

void checkCUDAError(const char *msg)
{
	hipError_t err = hipGetLastError();
	if (hipSuccess != err)
	{
		fprintf(stderr, "CUDA ERROR: %s: %s.\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}
